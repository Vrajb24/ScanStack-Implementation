#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

#include <vector>
#include <numeric>
#include <algorithm>
#include <set>
#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct {
    int* stack_data;
    int32_t stack_top;
    int stack_capacity;
} Stack;


__device__ bool push_gpu(Stack* stack, int value) {
    int32_t old_top;
    int32_t new_top;

    while (true) {
        old_top = atomicAdd(&stack->stack_top, 0);

        new_top = old_top + 1;

        if (new_top >= stack->stack_capacity) {
            return false;
        }

        int32_t assumed_old_top = atomicCAS((int32_t*)&stack->stack_top, old_top, new_top);

        if (assumed_old_top == old_top) {
            stack->stack_data[new_top] = value;
            return true;
        }
    }
}

__device__ bool pop_gpu(Stack* stack, int* result) {
    int32_t old_top;
    int32_t new_top;

    while (true) {
        old_top = atomicAdd(&stack->stack_top, 0);

        if (old_top < 0) {
            return false;
        }

        new_top = old_top - 1;

        int32_t assumed_old_top = atomicCAS((int32_t*)&stack->stack_top, old_top, new_top);

        if (assumed_old_top == old_top) {
            *result = stack->stack_data[old_top];
            return true;
        }
    }
}

__device__ bool peek_gpu(Stack* stack, int* result) {
    int32_t current_top = atomicAdd(&stack->stack_top, 0);

    if (current_top < 0) {
        return false;
    }

    *result = stack->stack_data[current_top];
    return true;
}


__global__ void stack_test_kernel(Stack* d_stack, int num_ops_total, bool* d_push_results, bool* d_pop_results, int* d_pop_values) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_ops_total) {
        if (tid % 2 == 0) {
            int value_to_push = tid;
            bool success = push_gpu(d_stack, value_to_push);
            if (d_push_results) d_push_results[tid / 2] = success;
        } else {
            int popped_value = -1;
            bool success = pop_gpu(d_stack, &popped_value);
            if (d_pop_results) d_pop_results[tid / 2] = success;
            if (d_pop_values && success) d_pop_values[tid / 2] = popped_value;
        }
    }
}


__global__ void reporting_kernel(Stack* d_stack, int num_ops, int base_value, int op_mode,
                                 int* d_op_tid, int* d_op_type, bool* d_op_success, int* d_op_value)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_ops) {
        d_op_tid[tid] = tid;
        d_op_type[tid] = op_mode;

        if (op_mode == 0) {
            int value_to_push = tid + base_value;
            bool success = push_gpu(d_stack, value_to_push);
            d_op_success[tid] = success;
            d_op_value[tid] = value_to_push;
        } else {
            int popped_value = -1;
            bool success = pop_gpu(d_stack, &popped_value);
            d_op_success[tid] = success;
            d_op_value[tid] = success ? popped_value : -1;
        }
    }
}

__global__ void push_check_success_kernel(Stack* d_stack, int num_pushes, int base_value, bool* d_push_results) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_pushes) {
        int value_to_push = tid + base_value;
        bool success = push_gpu(d_stack, value_to_push);
        if(d_push_results) d_push_results[tid] = success;
    }
}



void reset_stack(Stack* d_stack_struct_ptr, int* d_stack_data, int stack_capacity, bool clear_data = true) {
    int initial_top = -1;
    gpuErrchk(hipMemcpy(&(d_stack_struct_ptr->stack_top), &initial_top, sizeof(int32_t), hipMemcpyHostToDevice));
    if (clear_data) {
        gpuErrchk(hipMemset(d_stack_data, 0, stack_capacity * sizeof(int)));
    }
    gpuErrchk(hipDeviceSynchronize());
}

void print_operation_log(int num_ops, int* h_op_tid, int* h_op_type, bool* h_op_success, int* h_op_value) {
    printf("--- Operation Log ---\n");
    printf("Attempt | TID   | Op   | Value | Success\n");
    printf("--------|-------|------|-------|--------\n");
    for (int i = 0; i < num_ops; ++i) {
        printf("%-7d | %-5d | %-4s | %-5d | %s\n",
               i,
               h_op_tid[i],
               (h_op_type[i] == 0) ? "Push" : "Pop",
               h_op_value[i],
               h_op_success[i] ? "True" : "False");
    }
    printf("---------------------\n");
}



bool run_verification_pops_gt_pushes(Stack* d_stack_struct_ptr, int* d_stack_data, int stack_capacity, int block_size) {
    printf("\n--- Verification [1]: Pops > Pushes ---\n");
    bool test_passed = true;
    const int num_pushes = 10;
    const int num_pops = num_pushes + 5;
    const int base_value = 100;
    const int total_log_size = num_pushes + num_pops;
    printf("Config: Pushes=%d, Pops=%d, Capacity=%d\n", num_pushes, num_pops, stack_capacity);

    if (num_pushes > stack_capacity) {
        printf("Warning: num_pushes exceeds stack capacity for this test.\n");
    }

    int *d_op_tid, *h_op_tid;
    int *d_op_type, *h_op_type;
    bool *d_op_success, *h_op_success;
    int *d_op_value, *h_op_value;

    gpuErrchk(hipMalloc(&d_op_tid, total_log_size * sizeof(int)));
    gpuErrchk(hipMalloc(&d_op_type, total_log_size * sizeof(int)));
    gpuErrchk(hipMalloc(&d_op_success, total_log_size * sizeof(bool)));
    gpuErrchk(hipMalloc(&d_op_value, total_log_size * sizeof(int)));
    h_op_tid = (int*)malloc(total_log_size * sizeof(int));
    h_op_type = (int*)malloc(total_log_size * sizeof(int));
    h_op_success = (bool*)malloc(total_log_size * sizeof(bool));
    h_op_value = (int*)malloc(total_log_size * sizeof(int));

    reset_stack(d_stack_struct_ptr, d_stack_data, stack_capacity);

    printf("Running Push Phase...\n");
    int grid_size_push = (num_pushes + block_size - 1) / block_size;
    reporting_kernel<<<grid_size_push, block_size>>>(d_stack_struct_ptr, num_pushes, base_value, 0,
                                                    d_op_tid, d_op_type, d_op_success, d_op_value);
    gpuErrchk(hipDeviceSynchronize());

    printf("Running Pop Phase...\n");
    int grid_size_pop = (num_pops + block_size - 1) / block_size;
    reporting_kernel<<<grid_size_pop, block_size>>>(d_stack_struct_ptr, num_pops, base_value, 1,
                                                   d_op_tid + num_pushes, d_op_type + num_pushes,
                                                   d_op_success + num_pushes, d_op_value + num_pushes);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_op_tid, d_op_tid, total_log_size * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_type, d_op_type, total_log_size * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_success, d_op_success, total_log_size * sizeof(bool), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_value, d_op_value, total_log_size * sizeof(int), hipMemcpyDeviceToHost));

    print_operation_log(total_log_size, h_op_tid, h_op_type, h_op_success, h_op_value);

    int successful_pushes = 0;
    int successful_pops = 0;
    std::set<int> pushed_values_set;
    std::set<int> popped_values_set;

    for(int i = 0; i < total_log_size; ++i) {
        if (h_op_type[i] == 0) {
            if (h_op_success[i]) {
                successful_pushes++;
                pushed_values_set.insert(h_op_value[i]);
            }
        } else {
            if (h_op_success[i]) {
                successful_pops++;
                popped_values_set.insert(h_op_value[i]);
            }
        }
    }

    printf("Result: Successful Pushes: %d\n", successful_pushes);
    printf("Result: Successful Pops: %d (Expected <= %d)\n", successful_pops, successful_pushes);
    if (successful_pops > successful_pushes) {
        printf("FAIL: More pops succeeded than pushes!\n");
        test_passed = false;
    }
     if (successful_pops != successful_pushes) {
        printf("INFO: Number of successful pops (%d) doesn't exactly match successful pushes (%d). This might be ok if pushes failed or due to races.\n", successful_pops, successful_pushes);
     }


    int32_t final_top;
    gpuErrchk(hipMemcpy(&final_top, &(d_stack_struct_ptr->stack_top), sizeof(int32_t), hipMemcpyDeviceToHost));
    printf("Result: Final stack top: %d (Expected: -1)\n", final_top);
     if (final_top != -1) {
        printf("FAIL: Final stack top is not -1.\n");
        test_passed = false;
    }

    if (pushed_values_set != popped_values_set) {
         printf("FAIL: Set of popped values does not match set of successfully pushed values.\n");
         test_passed = false;
    }

    printf("Test Result: %s\n", test_passed ? "PASS" : "FAIL");

    free(h_op_tid); free(h_op_type); free(h_op_success); free(h_op_value);
    gpuErrchk(hipFree(d_op_tid)); gpuErrchk(hipFree(d_op_type)); gpuErrchk(hipFree(d_op_success)); gpuErrchk(hipFree(d_op_value));
    return test_passed;
}


bool run_verification_pop_empty(Stack* d_stack_struct_ptr, int* d_stack_data, int stack_capacity, int block_size) {
    printf("\n--- Verification [2]: Pop Empty Stack ---\n");
    bool test_passed = true;
    const int num_pops = 10;
    printf("Config: Pops=%d\n", num_pops);

    int *d_op_tid, *h_op_tid;
    int *d_op_type, *h_op_type;
    bool *d_op_success, *h_op_success;
    int *d_op_value, *h_op_value;

    gpuErrchk(hipMalloc(&d_op_tid, num_pops * sizeof(int)));
    gpuErrchk(hipMalloc(&d_op_type, num_pops * sizeof(int)));
    gpuErrchk(hipMalloc(&d_op_success, num_pops * sizeof(bool)));
    gpuErrchk(hipMalloc(&d_op_value, num_pops * sizeof(int)));
    h_op_tid = (int*)malloc(num_pops * sizeof(int));
    h_op_type = (int*)malloc(num_pops * sizeof(int));
    h_op_success = (bool*)malloc(num_pops * sizeof(bool));
    h_op_value = (int*)malloc(num_pops * sizeof(int));

    reset_stack(d_stack_struct_ptr, d_stack_data, stack_capacity);

    printf("Running Pop Phase...\n");
    int grid_size_pop = (num_pops + block_size - 1) / block_size;
    reporting_kernel<<<grid_size_pop, block_size>>>(d_stack_struct_ptr, num_pops, 0, 1,
                                                   d_op_tid, d_op_type, d_op_success, d_op_value);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_op_tid, d_op_tid, num_pops * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_type, d_op_type, num_pops * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_success, d_op_success, num_pops * sizeof(bool), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_op_value, d_op_value, num_pops * sizeof(int), hipMemcpyDeviceToHost));

    print_operation_log(num_pops, h_op_tid, h_op_type, h_op_success, h_op_value);

    int successful_pops = 0;
    for(int i = 0; i < num_pops; ++i) {
        if (h_op_success[i]) successful_pops++;
    }
    printf("Result: Successful pops: %d (Expected: %d)\n", successful_pops, 0);
    if (successful_pops != 0) {
        printf("FAIL: Pops succeeded on an empty stack.\n");
        test_passed = false;
    }

    int32_t final_top;
    gpuErrchk(hipMemcpy(&final_top, &(d_stack_struct_ptr->stack_top), sizeof(int32_t), hipMemcpyDeviceToHost));
    printf("Result: Final stack top: %d (Expected: %d)\n", final_top, -1);
     if (final_top != -1) {
        printf("FAIL: Final stack top changed.\n");
        test_passed = false;
    }

    printf("Test Result: %s\n", test_passed ? "PASS" : "FAIL");

    free(h_op_tid); free(h_op_type); free(h_op_success); free(h_op_value);
    gpuErrchk(hipFree(d_op_tid)); gpuErrchk(hipFree(d_op_type)); gpuErrchk(hipFree(d_op_success)); gpuErrchk(hipFree(d_op_value));
    return test_passed;
}

bool run_verification_overflow(Stack* d_stack_struct_ptr, int* d_stack_data, int stack_capacity, int block_size) {
    printf("\n--- Verification [3]: Stack Overflow ---\n");
    bool test_passed = true;
    const int test_capacity = 50;
    const int num_pushes = test_capacity + 10;
    const int base_value = 400;
    printf("Config: Test Capacity (Simulated)=%d, Pushes=%d\n", test_capacity, num_pushes);

    Stack h_stack_orig, h_stack_test;
    gpuErrchk(hipMemcpy(&h_stack_orig, d_stack_struct_ptr, sizeof(Stack), hipMemcpyDeviceToHost));
    h_stack_test = h_stack_orig;
    h_stack_test.stack_capacity = test_capacity;
    gpuErrchk(hipMemcpy(d_stack_struct_ptr, &h_stack_test, sizeof(Stack), hipMemcpyHostToDevice));

    bool *d_push_results, *h_push_results;
    gpuErrchk(hipMalloc(&d_push_results, num_pushes * sizeof(bool)));
    h_push_results = (bool*)malloc(num_pushes * sizeof(bool));

    reset_stack(d_stack_struct_ptr, d_stack_data, test_capacity, true);

    printf("Running Push Phase...\n");
    int grid_size_push = (num_pushes + block_size - 1) / block_size;
    push_check_success_kernel<<<grid_size_push, block_size>>>(d_stack_struct_ptr, num_pushes, base_value, d_push_results);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_push_results, d_push_results, num_pushes * sizeof(bool), hipMemcpyDeviceToHost));

    int successful_pushes = 0;
    for(int i = 0; i < num_pushes; ++i) {
        if (h_push_results[i]) successful_pushes++;
    }
    printf("Result: Successful pushes: %d (Expected: %d)\n", successful_pushes, test_capacity);
    if (successful_pushes != test_capacity) {
        printf("FAIL: Incorrect number of successful pushes during overflow.\n");
        test_passed = false;
    }

    int32_t final_top;
    gpuErrchk(hipMemcpy(&final_top, &(d_stack_struct_ptr->stack_top), sizeof(int32_t), hipMemcpyDeviceToHost));
    printf("Result: Final stack top: %d (Expected: %d)\n", final_top, test_capacity - 1);
     if (final_top != test_capacity - 1) {
        printf("FAIL: Final stack top is not capacity-1.\n");
        test_passed = false;
    }

    printf("Test Result: %s\n", test_passed ? "PASS" : "FAIL");

    free(h_push_results);
    gpuErrchk(hipFree(d_push_results));
    gpuErrchk(hipMemcpy(d_stack_struct_ptr, &h_stack_orig, sizeof(Stack), hipMemcpyHostToDevice));
    reset_stack(d_stack_struct_ptr, d_stack_data, stack_capacity, true);

    return test_passed;
}

int main() {
    int stack_capacity = 1024 * 100;
    const int block_size = 256;

    printf("Initializing Parallel Stack...\n");
    printf("Stack Capacity: %d\n", stack_capacity);
    printf("Block Size: %d\n", block_size);
    printf("========================================\n");

    int* d_stack_data;
    Stack* d_stack_struct_ptr;
    Stack h_stack;
    gpuErrchk(hipMalloc((void**)&d_stack_data, stack_capacity * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_stack_struct_ptr, sizeof(Stack)));
    h_stack.stack_data = d_stack_data;
    h_stack.stack_top = -1;
    h_stack.stack_capacity = stack_capacity;
    gpuErrchk(hipMemcpy(d_stack_struct_ptr, &h_stack, sizeof(Stack), hipMemcpyHostToDevice));

    printf("Starting Verification Tests...\n");
    bool all_tests_passed = true;


    all_tests_passed &= run_verification_pops_gt_pushes(d_stack_struct_ptr, d_stack_data, stack_capacity, block_size);
    all_tests_passed &= run_verification_pop_empty(d_stack_struct_ptr, d_stack_data, stack_capacity, block_size);
    all_tests_passed &= run_verification_overflow(d_stack_struct_ptr, d_stack_data, stack_capacity, block_size);


    printf("\n--- Overall Verification Result: %s ---\n", all_tests_passed ? "ALL PASS" : "SOME FAIL");
    printf("========================================\n");


    if (all_tests_passed) {
        printf("\nStarting Scalability Tests...\n");
        int num_ops_tests[] = {1000, 10000, 100000, 500000, 1000000, 2000000};

        for (int i = 0; i < sizeof(num_ops_tests) / sizeof(num_ops_tests[0]); ++i) {
            int num_operations = num_ops_tests[i];
            if (num_operations > stack_capacity * 10) {
                 printf("Skipping scalability test with %d operations: too large relative to capacity.\n", num_operations);
                 continue;
            }

            reset_stack(d_stack_struct_ptr, d_stack_data, stack_capacity);

            printf("\nRunning Test: %d Operations (Mixed Push/Pop)\n", num_operations);
            int grid_size = (num_operations + block_size - 1) / block_size;
            printf("Grid Size: %d\n", grid_size);

            hipEvent_t start, stop;
            gpuErrchk(hipEventCreate(&start)); gpuErrchk(hipEventCreate(&stop));
            gpuErrchk(hipEventRecord(start));
            stack_test_kernel<<<grid_size, block_size>>>(d_stack_struct_ptr, num_operations, nullptr, nullptr, nullptr);
            gpuErrchk(hipEventRecord(stop));
            gpuErrchk(hipGetLastError());
            gpuErrchk(hipEventSynchronize(stop));
            float milliseconds = 0;
            gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
            gpuErrchk(hipEventDestroy(start)); gpuErrchk(hipEventDestroy(stop));

            printf("Execution Time: %.4f ms\n", milliseconds);
            double seconds = milliseconds / 1000.0;
            if (seconds > 0) {
                double ops_per_second = (double)num_operations / seconds;
                printf("Throughput: %.2f Million Operations/Second\n", ops_per_second / 1e6);
            } else { printf("Throughput: Inf\n"); }
            printf("----------------------------------------");
        }
    } else {
         printf("\nSkipping Scalability Tests due to Verification Failures.\n");
    }

    printf("\nCleaning up...\n");
    gpuErrchk(hipFree(d_stack_data));
    gpuErrchk(hipFree(d_stack_struct_ptr));
    printf("Done.\n");
    return all_tests_passed ? 0 : 1;
}